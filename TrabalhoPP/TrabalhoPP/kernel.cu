#include "hip/hip_runtime.h"
﻿#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <hip/hip_runtime.h>
#include <>
#include <hiprand/hiprand_kernel.h>

#define TILE_SIZE 16 // Tamanho do bloco para memória compartilhada

__global__ void multiplicaMatrizCUDA(int* matriz1, int* matriz2, int* matrizResultado, int size) {
    __shared__ int tileM[TILE_SIZE][TILE_SIZE];
    __shared__ int tileN[TILE_SIZE][TILE_SIZE];

    int linha = blockIdx.y * TILE_SIZE + threadIdx.y;
    int coluna = blockIdx.x * TILE_SIZE + threadIdx.x;
    int valor = 0;

    for (int i = 0; i < (size + TILE_SIZE - 1) / TILE_SIZE; i++) {
        // Carregar dados na memória compartilhada
        if (linha < size && (i * TILE_SIZE + threadIdx.x) < size) {
            tileM[threadIdx.y][threadIdx.x] = matriz1[linha * size + (i * TILE_SIZE + threadIdx.x)];
        }
        else {
            tileM[threadIdx.y][threadIdx.x] = 0; // Preencher com 0 se fora dos limites
        }

        if (coluna < size && (i * TILE_SIZE + threadIdx.y) < size) {
            tileN[threadIdx.y][threadIdx.x] = matriz2[(i * TILE_SIZE + threadIdx.y) * size + coluna];
        }
        else {
            tileN[threadIdx.y][threadIdx.x] = 0; // Preencher com 0 se fora dos limites
        }

        __syncthreads(); // Sincroniza os threads

        // Multiplicação das matrizes
        for (int j = 0; j < TILE_SIZE; j++) {
            valor += tileM[threadIdx.y][j] * tileN[j][threadIdx.x];
        }

        __syncthreads(); // Sincroniza os threads novamente
    }

    if (linha < size && coluna < size) {
        matrizResultado[linha * size + coluna] = valor;
    }
}

// Kernel para preencher a matriz utilizando hiprand
__global__ void preencheMatrizCUDA(int* matriz, int size, unsigned long long seed) {
    int linha = blockIdx.y * blockDim.y + threadIdx.y;
    int coluna = blockIdx.x * blockDim.x + threadIdx.x;

    hiprandState state;
    hiprand_init(seed, linha * size + coluna, 0, &state); // Inicializa o gerador de números aleatórios

    if (linha < size && coluna < size) {
        matriz[linha * size + coluna] = hiprand(&state) % 10000; // Gera um número aleatório
    }
}

int main() {
    srand(time(0));
    int tamanho;
    printf("Insira o tamanho da matriz: ");
    scanf("%d", &tamanho);

    int* h_matriz1 = (int*)malloc(tamanho * tamanho * sizeof(int));
    int* h_matriz2 = (int*)malloc(tamanho * tamanho * sizeof(int));
    int* h_matrizResultado = (int*)malloc(tamanho * tamanho * sizeof(int));

    int* d_matriz1, * d_matriz2, * d_matrizResultado;
    hipMalloc((void**)&d_matriz1, tamanho * tamanho * sizeof(int));
    hipMalloc((void**)&d_matriz2, tamanho * tamanho * sizeof(int));
    hipMalloc((void**)&d_matrizResultado, tamanho * tamanho * sizeof(int));

    // Preenchendo a matriz 1 na GPU
    dim3 blocos(TILE_SIZE, TILE_SIZE);
    dim3 grades((tamanho + TILE_SIZE - 1) / TILE_SIZE, (tamanho + TILE_SIZE - 1) / TILE_SIZE);

    // Passando um valor de semente aleatório para cada chamada
    unsigned long long seed = time(0);
    preencheMatrizCUDA << <grades, blocos >> > (d_matriz1, tamanho, seed);
    preencheMatrizCUDA << <grades, blocos >> > (d_matriz2, tamanho, seed + 1);

    hipDeviceSynchronize(); // Espera os kernels terminarem

    // Copiando matrizes de volta para a CPU
    hipMemcpy(h_matriz1, d_matriz1, tamanho * tamanho * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(h_matriz2, d_matriz2, tamanho * tamanho * sizeof(int), hipMemcpyDeviceToHost);

    clock_t tempoInicial = clock();
    multiplicaMatrizCUDA << <grades, blocos >> > (d_matriz1, d_matriz2, d_matrizResultado, tamanho);
    hipDeviceSynchronize();
    clock_t tempoFinal = clock();

    double duracao = (double)(tempoFinal - tempoInicial) / CLOCKS_PER_SEC;
    printf("Tempo: %.2f clock/segundos\n", duracao);

    // Copiando o resultado da multiplicação para a CPU
    hipMemcpy(h_matrizResultado, d_matrizResultado, tamanho * tamanho * sizeof(int), hipMemcpyDeviceToHost);

    // Liberando memória
    free(h_matriz1);
    free(h_matriz2);
    free(h_matrizResultado);
    hipFree(d_matriz1);
    hipFree(d_matriz2);
    hipFree(d_matrizResultado);

    return 0;
}
